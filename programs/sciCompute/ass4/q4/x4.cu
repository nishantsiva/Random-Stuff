
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define SIZE 20

__global__ void max(int *input)
{
	__shared__ int s_data[SIZE];
	int tid = threadIdx.x, flag;
	int i = tid + blockIdx.x*blockDim.x;

	s_data[tid] = input[i];
	__syncthreads();

	for( int s = blockDim.x/2, flag = blockDim.x%2; s > 0; s = s >> 1)
	{
		if(tid < s) {
			printf("%d\t-\t%d\t%d\t%d\n", s_data[tid], s_data[tid+s], s, tid);
			s_data[tid] = (s_data[tid]>=s_data[tid+s]) ? s_data[tid] : s_data[tid+s];
		}	 
		__syncthreads();
		if(s == 1) break;
		flag = s%2;
	}
	if(tid == 0) input[0] = s_data[0];
}

int main(int argc, char *argv[])
{
	int *input;
	int *d_input;
	hipEvent_t start,stop;
	float etime;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	input = (int*)malloc(sizeof(int) * SIZE);
	
	srand(time(0));
	for(int j=0;j<SIZE;j++)
	{
		input[j] = rand()%SIZE;
		printf(" %d ",input[j]);
	}printf("\n");
	hipError_t err;
	err = hipMalloc((void**)&d_input,sizeof(int)*SIZE);
	if(err != hipSuccess)
		printf("Error allocating\n");

	//cudaMalloc((void**)&d_input,sizeof(int)*SIZE);
	hipMemcpy(d_input,input,sizeof(int)*SIZE, hipMemcpyHostToDevice);
	max<<<SIZE/10,10>>>(d_input);
	hipMemcpy(input,d_input,sizeof(int)*SIZE, hipMemcpyDeviceToHost);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&etime,start,stop);
	printf(" max:%d ",input[0]);
	printf("\nTime:%f\n",etime);
	return 0;
}

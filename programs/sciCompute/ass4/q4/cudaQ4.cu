
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define SIZE 1024*1024
#define nTHREAD 512

__global__ void min(float *input, float *out) {
	__shared__ float sData[nTHREAD];
	int tid = threadIdx.x;
	int iter = tid + blockIdx.x*blockDim.x;
	sData[tid] = input[iter];
	__syncthreads();
	for(int s = blockDim.x/2; s > 0; s /= 2) {
		if(iter < (s+blockIdx.x*blockDim.x)) {
			sData[tid] = (sData[tid] < sData[tid+s]) ? sData[tid] : sData[tid+s];  
		}
		__syncthreads();
	}
	if(tid == 0) {		
		out[blockIdx.x] = sData[0]; 
	}
}

__global__ void max(float *input, float *out) {
	__shared__ float sData[nTHREAD];
	int tid = threadIdx.x;
	int iter = tid + blockIdx.x*blockDim.x;
	sData[tid] = input[iter];
	__syncthreads();
	for(int s = blockDim.x/2; s > 0; s /= 2) {
		if(iter < (s+blockIdx.x*blockDim.x)) {
			sData[tid] = (sData[tid] > sData[tid+s]) ? sData[tid] : sData[tid+s];  
		}
		__syncthreads();
	}
	if(tid == 0) {		
		out[blockIdx.x] = sData[0]; 
	}
}

__global__ void sumTotal(float *input, float *out) {
	__shared__ float sData[nTHREAD];
	int tid = threadIdx.x;
	int iter = tid + blockIdx.x*blockDim.x;
	sData[tid] = input[iter];
	__syncthreads();
	for(int s = blockDim.x/2; s > 0; s /= 2) {
		if(iter < (s+blockIdx.x*blockDim.x)) {
			sData[tid] = sData[tid] + sData[tid+s];  
		}
		__syncthreads();
	}
	if(tid == 0) {		
		out[blockIdx.x] = sData[0]; 
	}
}

__global__ void subNsquare(float *input, float *out, float *mean) {
	__shared__ float sData[nTHREAD];
	int tid = threadIdx.x;
	int iter = tid + blockIdx.x*blockDim.x;
	sData[tid] = input[iter];
	out[iter] = pow(sData[tid] - *mean, 2);
}

__global__ void subNpowr4(float *input, float *out, float *mean) {
	__shared__ float sData[nTHREAD];
	int tid = threadIdx.x;
	int iter = tid + blockIdx.x*blockDim.x;
	sData[tid] = input[iter];
	out[iter] = pow(sData[tid] -* mean, 4); 
}

int main() {
	float *input, *out, *_input, *_out;
	hipEvent_t start, stop;
	float _time = 0.0;
	time_t seconds;	
	
	hipEventCreate(&start);	
	hipEventCreate(&stop);
	input = (float*)malloc(sizeof(float)*SIZE);
	out = (float*)malloc(sizeof(float)*SIZE);
	time(&seconds);
	srand((unsigned int)seconds);

	for(int i = 0; i < SIZE; i++) {
		input[i] = (float)rand()/(float)(RAND_MAX) * SIZE;
	}
	
	hipError_t err1, err2;
	err1 = hipMalloc((void**)&_input, sizeof(float)*SIZE);
	err2 = hipMalloc((void**)&_out, sizeof(float)*SIZE);
	if(err1 != hipSuccess || err2 != hipSuccess) {
		printf("Error in allocation %d %d\n", err1, err2);
		exit(0);
	}
	
	// Minimum Calculation
	hipMemcpy(_input, input, sizeof(float)*SIZE, hipMemcpyHostToDevice);
	
	hipEventRecord(start, 0);
	int flag = 0;
	int block = SIZE/nTHREAD, remainder;
	while(block > 1) {
		if(flag == 0) {
			min<<<block,nTHREAD >>>(_input, _out);
			flag = 1;
		}
		else {
			min<<<block,nTHREAD >>>(_out, _input);
			flag = 0;
		}
		remainder = block;
		block /= nTHREAD;
	}
	if(flag == 0) {
		min<<<1, remainder>>>(_input, _out);
		hipMemcpy(out, _out, sizeof(float)*SIZE, hipMemcpyDeviceToHost);
	}
	else {
		min<<<1, remainder>>>(_out, _input);	
		hipMemcpy(out, _input, sizeof(float)*SIZE, hipMemcpyDeviceToHost);
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&_time, start, stop);
	printf("Elapsed Time(MIN) = %f\n", _time);
	
	hipMemcpy(out, _out, sizeof(float)*SIZE, hipMemcpyDeviceToHost);
	printf("MIN = %f\n",out[0]);
	
	// Maximum Calculation
	hipMemcpy(_input, input, sizeof(float)*SIZE, hipMemcpyHostToDevice);
	
	hipEventRecord(start, 0);
	flag = 0;
	block = SIZE/nTHREAD;
	while(block > 1) {
		if(flag == 0) {
			max<<<block,nTHREAD >>>(_input, _out);
			flag = 1;
		}
		else {
			max<<<block,nTHREAD >>>(_out, _input);
			flag = 0;
		}
		remainder = block;
		block /= nTHREAD;
	}
	if(flag == 0) {
		max<<<1, remainder>>>(_input, _out);
		hipMemcpy(out, _out, sizeof(float)*SIZE, hipMemcpyDeviceToHost);
	}
	else {
		max<<<1, remainder>>>(_out, _input);	
		hipMemcpy(out, _input, sizeof(float)*SIZE, hipMemcpyDeviceToHost);
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&_time, start, stop);
	printf("Elapsed Time(MAX) = %.5f\n", _time);
	
	hipMemcpy(out, _out, sizeof(float)*SIZE, hipMemcpyDeviceToHost);
	printf("MAX = %f\n",out[0]);
	
	// STD calculation
	hipMemcpy(_input, input, sizeof(float)*SIZE, hipMemcpyHostToDevice);
	
	hipEventRecord(start, 0);
	flag = 0;
	block = SIZE/nTHREAD;
	while(block > 1) {
		if(flag == 0) {
			sumTotal<<<block, nTHREAD>>>(_input, _out);
			flag = 1;
		}
		else {
			sumTotal<<<block,nTHREAD >>>(_out, _input);
			flag = 0;
		}
		remainder = block;
		block /= nTHREAD;
	}
	if(flag == 0) {
		sumTotal<<<1, remainder>>>(_input, _out);
		hipMemcpy(out, _out, sizeof(float)*SIZE, hipMemcpyDeviceToHost);
	}
	else {
		sumTotal<<<1, remainder>>>(_out, _input);	
		hipMemcpy(out, _input, sizeof(float)*SIZE, hipMemcpyDeviceToHost);
	}
	
	float mean = out[0], *_mean = NULL;
	mean /= SIZE;
	hipMemcpy(_mean, &mean, sizeof(float), hipMemcpyHostToDevice);
	
	block = SIZE/nTHREAD;
	hipMemcpy(_input, input, sizeof(float)*SIZE, hipMemcpyHostToDevice);
	subNsquare<<<block, nTHREAD>>>(_input, _out, _mean);
	hipMemcpy(out, _out, sizeof(float)*SIZE, hipMemcpyDeviceToHost);	
	hipMemcpy(_input, out, sizeof(float)*SIZE, hipMemcpyHostToDevice);	
	flag = 0;
	block = SIZE/nTHREAD;
	while(block > 1) {
		if(flag == 0) {
			sumTotal<<<block, nTHREAD>>>(_input, _out);
			flag = 1;
		}
		else {
			sumTotal<<<block,nTHREAD >>>(_out, _input);
			flag = 0;
		}
		remainder = block;
		block /= nTHREAD;
	}
	if(flag == 0) {
		sumTotal<<<1, remainder>>>(_input, _out);
		hipMemcpy(out, _out, sizeof(float)*SIZE, hipMemcpyDeviceToHost);
	}
	else {
		sumTotal<<<1, remainder>>>(_out, _input);	
		hipMemcpy(out, _input, sizeof(float)*SIZE, hipMemcpyDeviceToHost);
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&_time, start, stop);
	printf("Elapsed Time(STD) = %.5f\n", _time);
	
	float std = out[0];
	std /= SIZE;
	std = sqrt(std);
	printf("STD = %f\n", std);
	
	// Kurtosis calculation

	hipMemcpy(_input, input, sizeof(float)*SIZE, hipMemcpyHostToDevice);
	
	hipEventRecord(start, 0);
	block = SIZE/nTHREAD;
	subNpowr4<<<block, nTHREAD>>>(_input, _out, _mean);

	hipMemcpy(out, _out, sizeof(float)*SIZE, hipMemcpyDeviceToHost);	
	hipMemcpy(_input, out, sizeof(float)*SIZE, hipMemcpyHostToDevice);	
	flag = 0;
	block = SIZE/nTHREAD;
	while(block > 1) {
		if(flag == 0) {
			sumTotal<<<block, nTHREAD>>>(_input, _out);
			flag = 1;
		}
		else {
			sumTotal<<<block,nTHREAD >>>(_out, _input);
			flag = 0;
		}
		remainder = block;
		block /= nTHREAD;
	}
	if(flag == 0) {
		sumTotal<<<1, remainder>>>(_input, _out);
		hipMemcpy(out, _out, sizeof(float)*SIZE, hipMemcpyDeviceToHost);
	}
	else {
		sumTotal<<<1, remainder>>>(_out, _input);	
		hipMemcpy(out, _input, sizeof(float)*SIZE, hipMemcpyDeviceToHost);
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&_time, start, stop);
	printf("Elapsed Time(kurtosis) = %.5f\n", _time);
	
	float kurtosis = out[0];
	kurtosis /= SIZE;
	kurtosis /= pow(std, 4);
	printf("Kurtosis = %f\n", kurtosis);

	hipFree(_input);
	hipFree(_out);
	hipFree(_mean);
}
